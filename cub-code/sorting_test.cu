#include "hip/hip_runtime.h"
//#include "../../cub-1.8.0/hipcub/hipcub.hpp"   // or equivalently <cub/device/device_histogram.cuh>
#include "cub.cuh"
#include "helper.cu.h"
#include "kernels.cu.h"
#include <math.h>

#define blockMemSize 1024

void printBits(int val){
    for(unsigned int mask = 0x80000000; mask; mask >>= 1){
         printf("%d", !!(mask & val));
    }
    printf("\n");
}

template<class Z>
bool validateZ(Z* A, uint32_t sizeAB) {
    for(uint32_t i = 1; i < sizeAB; i++)
      if (A[i-1] > A[i]){
        printf("INVALID RESULT for i:%d, (A[i-1]=%d > A[i]=%d)\n", i, A[i-1], A[i]);
        return false;
      }
    return true;
}

void randomInitNat(uint32_t* data, const uint32_t size, const uint32_t H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand()%16;
        data[i] = r % H;
        
    }
}

double sortRedByKeyCUB( uint32_t* data_keys_in
                      , uint32_t* data_keys_out
                      , const uint64_t N
) {
    int beg_bit = 0;
    int end_bit = 32;

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;

    { // sort prelude
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }
    cudaCheckError();

    { // one dry run
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
    }
    hipDeviceSynchronize();
    cudaCheckError();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);

    return elapsed;
}


int main (int argc, char * argv[]) {
    if (argc != 2) {
        printf("Usage: %s <size-of-array>\n", argv[0]);
        exit(1);
    }
    const uint64_t N = atoi(argv[1]);

    //Allocate and Initialize Host data with random values
    uint32_t* h_keys  = (uint32_t*) malloc(N*sizeof(uint32_t));
    uint32_t* h_keys_res  = (uint32_t*) malloc(N*sizeof(uint32_t));
    randomInitNat(h_keys, N, N/10);

    //Allocate and Initialize Device data
    uint32_t* d_keys_in;
    uint32_t* d_keys_out;
    cudaSucceeded(hipMalloc((void**) &d_keys_in,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_keys_in, h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &d_keys_out, N * sizeof(uint32_t)));

    double elapsed = sortRedByKeyCUB( d_keys_in, d_keys_out, N );

    hipMemcpy(h_keys_res, d_keys_out, N*sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();

    bool success = validateZ(h_keys_res, N);

    printf("CUB Sorting for N=%lu runs in: %.2f us, VALID: %d\n", N, elapsed, success);

    // Cleanup and closing
    hipFree(d_keys_in); hipFree(d_keys_out);
    free(h_keys); free(h_keys_res);


    //  ** New kernel section ** 
    // setup execution parameters
    int dimbl = 1;//(int) (sqrt(ceil(N/1024))) + 1;
    dim3 block(32, 32, 1); // 256 threads per block
    dim3 grid (dimbl, dimbl, 1); 

    //Allocate and Initialize Host data with random values
    
    uint32_t* global_histogram_output  = (uint32_t*) malloc(dimbl * dimbl * 16 *sizeof(uint32_t)); // todo:fix size, but who cares
    uint32_t* keys  = (uint32_t*) malloc(N*sizeof(uint32_t));
    uint32_t* keys_res  = (uint32_t*) malloc(N*sizeof(uint32_t));
    randomInitNat(keys, N, N/10);

    //Allocate and Initialize Device data
    uint32_t* keys_in;
    uint32_t* keys_sort;
    uint32_t* keys_out;
    uint32_t* glb_bins;
    uint32_t* scanned_glb_bins;

    uint32_t num_glb_bins = dimbl * dimbl * 16 * sizeof(uint32_t);
    cudaSucceeded(hipMalloc((void**) &keys_in,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(keys_in, keys, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &keys_sort,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMalloc((void**) &keys_out, N * sizeof(uint32_t)));

    cudaSucceeded(hipMalloc((void**) &glb_bins, num_glb_bins));
    cudaSucceeded(hipMalloc((void**) &scanned_glb_bins, num_glb_bins));
    hipMemset(glb_bins, 0, dimbl * dimbl * 16 * sizeof(uint32_t));
    hipMemset(scanned_glb_bins, 0, dimbl * dimbl * 16 * sizeof(uint32_t));

    //    double elapsed = sortRedByKeyCUB( keys_in, deys_out, N );
    double elapsedKernel;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    // Initialize vars for devicescan
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, glb_bins, scanned_glb_bins, num_glb_bins);  
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    for(int q=0; q<GPU_RUNS; q++) {
        for (int iter=0; iter<1; iter++){
            kern1<blockMemSize><<< grid, block >>>(keys_in, keys_out, glb_bins, N ,iter);
            //kern3<blockMemSize><<< grid, block >>>(glb_bins, scanned_glb_bins, num_glb_bins);
            //hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, glb_bins, scanned_glb_bins, num_glb_bins);	
            //kern4<blockMemSize><<< grid, block >>>(scanned_glb_bins, keys_out, keys_sort, N ,iter, glb_bins);
            //keys_in = keys_sort;
        }
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsedKernel = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);


    hipMemcpy(keys_res, keys_out, N*sizeof(uint32_t), hipMemcpyDeviceToHost); // todo: fix keys_in
    hipDeviceSynchronize();
    cudaCheckError();
    
    
    // for (size_t i = 0; i < N; i++)
    //   {
	//     printf("%d\n",keys_res[i]);
    //     //printBits(keys_res[i]);
    //   }
    
    hipMemcpy(global_histogram_output, glb_bins, dimbl * dimbl* 16 *sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();
    


    bool successKernel = validateZ(keys_res, N);

    printf("Our sorting for N=%lu runs in: %.2f us, VALID: %d\n", N, elapsedKernel, successKernel);

    // Cleanup and closing
    hipFree(keys_in); hipFree(keys_out); hipFree(keys_sort); hipFree(glb_bins); hipFree(scanned_glb_bins);
    hipFree(d_temp_storage);
    free(keys); free(keys_res);


    return success ? 0 : 1;

}
